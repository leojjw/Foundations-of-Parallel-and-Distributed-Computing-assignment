#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "SpMV.hpp"
#include "VectorOperator.hpp"

const int threads_per_block = 256;

__global__
void gpu_SpMV_kernel(double* d_A, double* d_V, double* d_result, unsigned int M, unsigned int Xnode) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i == 0 || i >= M) return;
  d_result[i] = d_A[i] * d_V[i] + d_A[M + i] * d_V[i - 1 + Xnode] +
                d_A[2 * M + i] * d_V[i + Xnode] + d_A[3 * M + i] * d_V[i + 1 + Xnode] +
                d_A[4 * M + i] * d_V[i + Xnode + Xnode];
}

void gpu_SpMV(double** A, double* V, double* result, unsigned int M, unsigned int Xnode) {
  int size = M * sizeof(double);
  double *d_A, *d_V, *d_result;

  hipMalloc((void **) &d_A, 5 * size);
  hipMalloc((void **) &d_V, size + 2 * Xnode * sizeof(double));
  hipMalloc((void **) &d_result, size);
  for (int i = 0; i < 5; i++){
    hipMemcpy(d_A + M * i, A[i], size, hipMemcpyHostToDevice);
  }
  hipMemcpy(d_V, V, size + 2 * Xnode * sizeof(double), hipMemcpyHostToDevice);

  dim3 grid_dim(ceil((double)M/threads_per_block), 1, 1);
  dim3 block_dim(threads_per_block, 1, 1);
  gpu_SpMV_kernel<<<grid_dim, block_dim>>>(d_A, d_V, d_result, M, Xnode);

  hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);
  hipFree(d_A); hipFree(d_V); hipFree(d_result);
}

__global__
void gpu_VectorDotVector_kernel(double* d_a, double* d_b, double* d_sum, unsigned int L) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= L) return;
  d_sum[i] = d_a[i] * d_b[i];
}

double gpu_VectorDotVector(double* a, double* b, unsigned int L) {
  int size = L * sizeof(double);
  double *d_a, *d_b, *d_sum, *sum;
  sum = new double[L];

  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_b, size);
  hipMalloc((void **) &d_sum, size);
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  dim3 grid_dim(ceil((double)L/threads_per_block), 1, 1);
  dim3 block_dim(threads_per_block, 1, 1);
  gpu_VectorDotVector_kernel<<<grid_dim, block_dim>>>(d_a, d_b, d_sum, L);
  hipDeviceSynchronize();

  hipMemcpy(sum, d_sum, size, hipMemcpyDeviceToHost);

  double result = 0;
  for (int i = 0; i < L; i++) {
    result += sum[i];
  }
  hipFree(d_a); hipFree(d_b); hipFree(d_sum);
  delete[] sum;

  return result;
}

double gpu_VectorNorm(double* a, unsigned int L) {
  return gpu_VectorDotVector(a, a, L);
}

__device__
void lock(int* mutex) {
  while (atomicCAS(mutex, 0, 1) != 0);
}

__device__
void unlock(int* mutex) {
  atomicExch(mutex, 0);
}

__global__
void gpu_VectorDotVector_optimized_kernel(double* d_a, double* d_b, double* d_sum, unsigned int L, int* mutex) {
  __shared__ double thread_sum[threads_per_block];
  
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx >= L) return;
  thread_sum[threadIdx.x] = d_a[idx] * d_b[idx];
  __syncthreads();

  int i = threads_per_block / 2;
  while(i != 0) {
    if (threadIdx.x < i && idx + i < L) {
      thread_sum[threadIdx.x] += thread_sum[threadIdx.x + i];
    }
    __syncthreads();
    i = i / 2;
  }

  if (threadIdx.x == 0) {
    lock(mutex);
    *d_sum += thread_sum[0];
    unlock(mutex);
  }
}

double gpu_VectorDotVector_optimized(double* a, double* b, unsigned int L) {
  int size = L * sizeof(double);
  double sum = 0;
  double *d_a, *d_b, *d_sum;

  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_b, size);
  hipMalloc((void **) &d_sum, sizeof(double));
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  hipMemcpy(d_sum, &sum, sizeof(double), hipMemcpyHostToDevice);
  
  int *mutex = NULL;
  hipMalloc((void **)&mutex, sizeof(int));
  hipMemset(mutex, 0, sizeof(int));

  dim3 grid_dim(ceil((double)L/threads_per_block), 1, 1);
  dim3 block_dim(threads_per_block, 1, 1);
  gpu_VectorDotVector_optimized_kernel<<<grid_dim, block_dim>>>(d_a, d_b, d_sum, L, mutex);
  hipDeviceSynchronize();
  hipMemcpy(&sum, d_sum, sizeof(double), hipMemcpyDeviceToHost);
  hipFree(d_a); hipFree(d_b); hipFree(d_sum); hipFree(mutex);

  return sum;
}

double gpu_VectorNorm_optimized(double* a, unsigned int L) {
  return gpu_VectorDotVector(a, a, L);
}